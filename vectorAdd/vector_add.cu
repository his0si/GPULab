#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  vector_add.cu
 *
 *    Description: Vector Addition
 *
 *        Version:  1.0
 *        Created:  07/08/2021 09:54:25 AM
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Myung Kuk Yoon 
 *   Organization:  EWHA Womans University
 *
 * =====================================================================================
 */

/* 
 * There are five exercises where some code is missing. Please fill in the missing code to complete the vector addition program
 */

//Exercise #05: Change MAX_SIZE to see how the performance changes
const int MAX_SIZE=10000; 
const float MAX_NUM=100.0;
const int MAX_ITER= 100;

#include <iostream>
#include "clockMeasure.h"

float inputA[MAX_SIZE];
float inputB[MAX_SIZE];
float gpuAns[MAX_SIZE];
float cpuAns[MAX_SIZE];

using namespace std;

#define checkCudaError(error) 			\
	if(error != hipSuccess){ 				\
		printf("%s in %s at line %d\n", \
				hipGetErrorString(error), 	\
				__FILE__ ,__LINE__); 				\
		exit(EXIT_FAILURE);							\
	}

void generateRandomValues(float *array, float max, const int size){
	for(int i = 0; i < size; i++){
		array[i] = float(rand())/float(RAND_MAX) * max;
	}
}

void cpuVectorAddition(float *h_a, float *h_b, float *h_c, const int size){
	for(int i = 0; i < size; i++){
		h_c[i] = h_a[i] + h_b[i];
	}
}
//Exercise #04: Insert vector addition code
__global__
void gpuVectorAddition(float *d_a, float *d_b, float *d_c, const int size){
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	if(tId < size){
		d_c[tId] = d_a[tId] + d_b[tId];
		}
		}
void checkAnswer(float *h_a, float *d_a, const int size){
	bool isSame = true;
	for(int i = 0; i < size; i++){
		if(h_a[i] != d_a[i]){
			cout<<"-\tERROR: IDX - "<< i << " (" << h_a[i] << " != " << d_a[i] << " )" << endl;
			isSame = false;
		}
	}
	if(isSame)
		printf("All values are same\n");
	else
		printf("Some values are not same\n");
}

//Main Function
int main(){
	srand((unsigned int)time(NULL));

	clockMeasure *ckCpu = new clockMeasure("CPU CODE");
	ckCpu->clockReset();
	clockMeasure *ckGpu = new clockMeasure("GPU CODE");
	ckGpu->clockReset();
	
	//Random
	generateRandomValues(inputA, MAX_NUM, MAX_SIZE);
	generateRandomValues(inputB, MAX_NUM, MAX_SIZE);
	
	//GPU Memory
	float *d_a, *d_b, *d_c;
	int arraySize = MAX_SIZE * sizeof(float);
	hipError_t err = hipMalloc((void **) &d_a, arraySize);
	checkCudaError(err);
	err = hipMalloc((void **) &d_b, arraySize);
	checkCudaError(err);
	err = hipMalloc((void **) &d_c, arraySize);
	checkCudaError(err);
	//Exercise #01: Allocate memory on GPUs for d_b and d_c

	err = hipMemcpy(d_a, inputA, arraySize, hipMemcpyHostToDevice);
	checkCudaError(err);
	err = hipMemcpy(d_b, inputB, arraySize, hipMemcpyHostToDevice);
	checkCudaError(err);
	//Exercise #02: Copy data of inputB to d_b

	const int tSize = 256;
	dim3 gridSize(ceil((float)MAX_SIZE/(float)tSize), 1, 1);
	dim3 blockSize(tSize, 1, 1);
	
	for(int i = 0; i < MAX_ITER; i++){
		ckCpu->clockResume();
		cpuVectorAddition(inputA, inputB, cpuAns, MAX_SIZE);
		ckCpu->clockPause();
		
		ckGpu->clockResume();
		//Exercise #03: Call Kernel Function
		gpuVectorAddition<<<gridSize, blockSize>>>(d_a, d_b, d_c, MAX_SIZE);

		err=hipDeviceSynchronize();
		ckGpu->clockPause();
		checkCudaError(err);
	}

	err = hipMemcpy(gpuAns, d_c, arraySize, hipMemcpyDeviceToHost);
	checkCudaError(err);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	checkAnswer(cpuAns, gpuAns, MAX_SIZE);

	ckCpu->clockPrint();
	ckGpu->clockPrint();

	return 0;
}
